#include <opencv2/videoio.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

#include <iostream>
#include <string>
#include <vector>
#include <chrono>

#include "kernels.cuh"

using namespace cv; 
using namespace std; 

// Timing Macros
#define MEASURE_GPU_TIME(func, result) \
do { \
    hipEvent_t startEvent, stopEvent; \
    hipEventCreate(&startEvent); \
    hipEventCreate(&stopEvent); \
    hipEventRecord(startEvent); \
    func; \
    hipEventRecord(stopEvent); \
    hipEventSynchronize(stopEvent); \
    float milliseconds = 0; \
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent); \
    result = static_cast<double>(milliseconds); \
    hipEventDestroy(startEvent); \
    hipEventDestroy(stopEvent); \
    } while (0)

struct Parameters {
    float phase_strength; 
    float warp_strength; 
    float spectral_phase_variance;
    float regularization_term; 
    float phase_activation_gain; 

    Parameters() : phase_strength(1), warp_strength(1), spectral_phase_variance(1), 
        regularization_term(1), phase_activation_gain(1)
    {}
};

int frameCount = 0; 
chrono::high_resolution_clock::time_point startTime, endTime;

// Function to display the achieved framerate
void displayFramerate(Mat& image, double fps) {
    string text = "FPS: " + to_string(fps);
    int fontFace = FONT_HERSHEY_SIMPLEX;
    double fontScale = 1;
    int thickness = 2;
    int baseline = 0;

    Size textSize = getTextSize(text, fontFace, fontScale, thickness, &baseline);
    Point textOrg(image.cols - textSize.width - 10, image.rows - 10);

    rectangle(image, textOrg + Point(0, baseline), textOrg + Point(textSize.width, -textSize.height), Scalar(0, 0, 0), FILLED);
    putText(image, text, textOrg, fontFace, fontScale, Scalar(255, 255, 255), thickness);
}


int main(int argc, char** argv) {

    //string GSTREAMER_PIPELINE = "nvarguscamerasrc sensor-mode=0   exposuretimerange='1000000 1000000' wbmode=0 gainrange='1 1' ispdigitalgainrange='1 1' tnr-mode=2 tnr-strength=1.0 ee-mode=2 ! video/x-raw(memory:NVMM), width=(int)1280, height=(int)720,format=(string)NV12, framerate=(fraction)21/1 ! nvvidconv ! video/x-raw, format=(string)BGRx ! videoconvert !  appsink";

    // Set up frame buffer and VideoCapture object
    Mat image;
    VideoCapture cap(CAP_ANY);
    if (!cap.isOpened()) {
        cout << "No video stream detected." << endl;
        system("pause");
        return 1;
    }

    // Initialize and set Frame Width And Height
    const size_t width = 1920; 
    const size_t height = 1080; 
    const size_t N = width * height; 
    cap.set(CAP_PROP_FRAME_WIDTH, width); 
    cap.set(CAP_PROP_FRAME_HEIGHT, height);
    cout << "width: " << width << endl;
    cout << "height: " << height << endl;

    namedWindow("Raw Feed", WINDOW_NORMAL); 
    namedWindow("VEViD", WINDOW_NORMAL); 

    double averageFramerate = 0.0; 

    // Allocate GPU memory
    hipfftComplex* d_vevid_kernel;
    hipfftComplex* d_image; 
    uint8_t* d_buffer; 
    float* d_max; 
    float* d_min; 
    hipMalloc((void**)&d_vevid_kernel, N * sizeof(hipfftComplex)); 
    hipMalloc((void**)&d_image, N * sizeof(hipfftComplex)); 
    hipMalloc((void**)&d_buffer, N * sizeof(uint8_t)); 
    hipMalloc((void**)&d_max, sizeof(float));
    hipMalloc((void**)&d_min, sizeof(float));

    // Initialize FFT Plans
    hipfftHandle plan; 
    hipfftPlan2d(&plan, (int)height, (int)width, HIPFFT_C2C); // swapped width and height because cufft requires column-major order

    // Set up input parameters
    Parameters params;
    params.phase_strength = 10;
    params.spectral_phase_variance = 0.1; 
    params.regularization_term = 4; 
    params.phase_activation_gain = 2.2; 

    // Measure GPU Operations Time
    float HtoD_time = 0;
    float FFT_time = 0; 
    float vevid_time = 0; 
    float max_time = 0; 
    float FFTShift_time = 0;
    float hadamard_time = 0; 
    float IFFT_time = 0; 
    float phase_time = 0;
    float DtoH_time = 0; 

    // Measure CPU Operations Time
    float total_time = 0; 
    float read_time = 0; 
    float BGRtoHSV_time = 0; 
    float merge_time = 0; 
    float HSVtoBGR_time = 0; 

    double componentTime = 0; 

    // Set up while loop to read video frames into buffer
    while (true) {

        startTime = chrono::high_resolution_clock::now();
        // Read frame into buffer
        auto startRead = chrono::high_resolution_clock::now(); 
	    cap >> image; 
        auto endRead = chrono::high_resolution_clock::now(); 
        chrono::duration<float> read_frame = chrono::duration_cast<chrono::duration<float>>(endRead - startRead);
        read_time += read_frame.count(); 

	    // Display unaltered video feed (for reference)
        imshow("Raw Feed", image);

        // Convert from BGR to HSV 
        auto startBGRtoHSV = chrono::high_resolution_clock::now(); 
        cvtColor(image, image, COLOR_BGR2HSV); 
        auto endBGRtoHSV = chrono::high_resolution_clock::now(); 
        chrono::duration<float> BGRtoHSV_frame = chrono::duration_cast<chrono::duration<float>>(endBGRtoHSV - startBGRtoHSV);
        BGRtoHSV_time += BGRtoHSV_frame.count(); 

        // Split channels of HSV matrix
        vector<Mat> hsv_channels; 
        split(image, hsv_channels);  

        // Get pointer to V channel of HSV matrix
        uint8_t* idata = hsv_channels[2].ptr<uint8_t>(0);

        // --Start of Algorithm Code--
        // Copy data from host to device
        MEASURE_GPU_TIME(hipMemcpy(d_buffer, idata, N * sizeof(uint8_t), hipMemcpyHostToDevice), componentTime); 
        HtoD_time += componentTime;  

        // Call kernels
        int block_size = 32; 
        int grid_size = ((int)N + block_size - 1) / block_size;

        MEASURE_GPU_TIME((populate_real<<<grid_size, block_size>>>(d_image, d_buffer, N)), componentTime); 
        FFT_time += componentTime; 
        MEASURE_GPU_TIME((add <<<grid_size, block_size>>> (d_image, params.regularization_term, N)), componentTime); 
        FFT_time += componentTime; 
        MEASURE_GPU_TIME(hipfftExecC2C(plan, d_image, d_image, HIPFFT_FORWARD), componentTime); 
        FFT_time += componentTime; 

        // Compute Kernel
        MEASURE_GPU_TIME((vevid_kernel <<<grid_size, block_size>>> (d_vevid_kernel, params.phase_strength, params.spectral_phase_variance, width, height)), componentTime); 
        vevid_time += componentTime; 
        MEASURE_GPU_TIME((max_reduce <<<64, block_size, block_size * sizeof(float)>>> (d_vevid_kernel, d_max, N)), componentTime); 
        vevid_time += componentTime; 

        // FFTShift
        float max_val; 
        MEASURE_GPU_TIME((hipMemcpy(&max_val, d_max, sizeof(float), hipMemcpyDeviceToHost)), componentTime); 
        FFTShift_time += componentTime; 
        MEASURE_GPU_TIME((scale <<<grid_size, block_size>>> (d_vevid_kernel, (1.0f / max_val), N)), componentTime); 
        FFTShift_time += componentTime; 
        MEASURE_GPU_TIME((fftshift <<<grid_size, block_size>>> (d_vevid_kernel, width, height)), componentTime); 
        FFTShift_time += componentTime; 

        // Multiply image with vevid kernel in frequency domain
        MEASURE_GPU_TIME((hadamard <<<grid_size, block_size>>> (d_vevid_kernel, d_image, N)), componentTime); 
        hadamard_time += componentTime; 

        // Take IFFT
        MEASURE_GPU_TIME(hipfftExecC2C(plan, d_image, d_image, HIPFFT_BACKWARD), componentTime); 
        IFFT_time += componentTime; 
        MEASURE_GPU_TIME((scale <<<grid_size, block_size>>> (d_image, (1.0f / (float)N), N)), componentTime); 
        IFFT_time += componentTime; 

        // Get vevid phase
        MEASURE_GPU_TIME((vevid_phase <<<grid_size, block_size>>> (d_image, d_buffer, params.phase_activation_gain, N)), componentTime); 
        phase_time += componentTime; 
	    // --End of Algorithm Code

        // Copy data from device to host
        MEASURE_GPU_TIME(hipMemcpy(idata, d_buffer, N * sizeof(uint8_t), hipMemcpyDeviceToHost), componentTime); 
        DtoH_time += componentTime; 
       	
        auto startMerge = chrono::high_resolution_clock::now(); 
        merge(hsv_channels, image); 
        auto endMerge = chrono::high_resolution_clock::now(); 
        chrono::duration<float> merge_frame = chrono::duration_cast<chrono::duration<double>>(endMerge - startMerge); 
        merge_time += merge_frame.count(); 

        // Convert from HSV to BGR
        auto startHSVtoBGR = chrono::high_resolution_clock::now(); 
        cvtColor(image, image, COLOR_HSV2BGR); 
        auto endHSVtoBGR = chrono::high_resolution_clock::now(); 
        chrono::duration<float> HSVtoBGR_frame = chrono::duration_cast<chrono::duration<double>>(endHSVtoBGR - startHSVtoBGR); 
        HSVtoBGR_time += HSVtoBGR_frame.count();

        // Calculate and display the achieved framerate
        frameCount++; 
        endTime = chrono::high_resolution_clock::now(); 
        chrono::duration<double> duration = chrono::duration_cast<chrono::duration<double>>(endTime - startTime);
        total_time += duration.count(); 
        double currentFramerate = frameCount / duration.count();
        averageFramerate = (averageFramerate * (frameCount - 1) + currentFramerate) / frameCount;
        displayFramerate(image, averageFramerate);

        // Display Processed video feed
        imshow("VEViD", image); 

        // Exit on escape key
        char c = (char)waitKey(1);
        if (c == 27) {
            float averageHtoDCopyTimePerFrame = HtoD_time / frameCount; 
            float averageFFTTimePerFrame = FFT_time / frameCount; 
            float averageVevidTimePerFrame = vevid_time / frameCount; 
            float averageMaxTimePerFrame = max_time / frameCount; 
            float averageFFTShiftTimePerFrame = FFTShift_time / frameCount; 
            float averageHadamardTimePerFrame = hadamard_time / frameCount; 
            float averageIFFTTimePerFrame = IFFT_time / frameCount; 
            float averagePhaseTimePerFrame = phase_time / frameCount; 
            float averageDtoHCopyTimePerFrame = DtoH_time / frameCount; 

            float averageTime = total_time / frameCount; 
            float averageReadTimePerFrame = read_time /frameCount; 
            float averageBGRtoHSVTimePerFrame = BGRtoHSV_time / frameCount; 
            float averageMergeTimePerFrame = merge_time / frameCount; 
            float averageHSVtoBGRTimePerFrame = HSVtoBGR_time / frameCount; 

            cout << "Frames Captured: " << frameCount << endl; 
            cout << "Total Time: " << averageTime << endl; 
            cout << "--Gpu Operations--" << endl; 
            cout << "Average Host to Device Copy Time Per Frame: " << averageHtoDCopyTimePerFrame << " ms" << endl; 
            cout << "Average FFT Time Per Frame: " << averageFFTTimePerFrame << " ms" << endl; 
            cout << "Average Vevid Kernel Time Per Frame: " << averageVevidTimePerFrame << " ms" << endl; 
            cout << "Average Max Reduce Time Per Frame: " << averageMaxTimePerFrame << " ms" << endl; 
            cout << "Average FFTShift Time Per Frame: " << averageFFTShiftTimePerFrame << " ms" << endl; 
            cout << "Average Hadamard Product Time Per Frame: " << averageHadamardTimePerFrame << " ms" << endl; 
            cout << "Average IFFT Time Per Frame: " << averageIFFTTimePerFrame << " ms" << endl; 
            cout << "Average Phase Time Per Frame: " << averagePhaseTimePerFrame << " ms" << endl; 
            cout << "Average Device to Host Copy Time Per Frame: " << averageDtoHCopyTimePerFrame << " ms" << endl; 
            cout << "--CPU Operations--" << endl; 
            cout << "Average Read Time Per Frame: " << averageReadTimePerFrame * 1000 << " ms" << endl; 
            cout << "Average BGR to HSV Conversion Time Per Frame: " << averageBGRtoHSVTimePerFrame * 1000 << " ms" << endl; 
            cout << "Average Merge Time Per Frame: " << averageMergeTimePerFrame * 1000 << " ms" << endl; 
            cout << "Average HSV to BGR Conversion Time Per Frame: " << averageHSVtoBGRTimePerFrame * 1000 << " ms" << endl; 

            break;
        }
    }
    // Free frame buffer
    cap.release(); 

    // Free GPU memory
    hipFree(d_vevid_kernel); 
    hipFree(d_image); 
    hipFree(d_buffer);
    hipFree(d_max); 
    hipFree(d_min); 

    return 0; 
}

