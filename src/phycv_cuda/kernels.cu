#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>

#include <stdint.h>
#include <float.h>
#include <stdio.h>

__global__
void init_kernel(hipfftComplex* d_vevid_kernel, float S, float T, int width, int height)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < width * height; i += stride) {
        int row = i / width;
        int col = i % width;
        float u = -0.5 + ((0.5 + 0.5) / (height - 1)) * floorf((row * width + col) / width);
        float v = -0.5 + ((0.5 + 0.5) / (width - 1)) * col;
        float value = sqrtf((u * u) + (v * v));
        float x = expf(-(value * value) / T);
        d_vevid_kernel[i].x = x; 
    }
}

__device__ 
float atomicMaxf(float* address, float val)
{
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
            __float_as_int(val));
    }
    return __int_as_float(old);
}

__device__ 
float atomicMinf(float* address, float val)
{
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;
    while (val < __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
            __float_as_int(val));
    }
    return __int_as_float(old);
}


__global__ 
void max_reduce(const hipfftComplex* const d_array, float* d_max, const size_t N)
{
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int gid = (blockDim.x * blockIdx.x) + tid;
    shared[tid] = -FLT_MAX;

    while (gid < N) {
        shared[tid] = max(shared[tid], d_array[gid].x);
        gid += gridDim.x * blockDim.x;
    }
    __syncthreads();
    gid = (blockDim.x * blockIdx.x) + tid;  
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s && gid < N)
            shared[tid] = max(shared[tid], shared[tid + s]);
        __syncthreads();
    }

    if (tid == 0)
        atomicMaxf(d_max, shared[0]);
}

__global__ 
void min_max_reduce(hipfftComplex* d_array, float* d_max, float* d_min, const size_t N)
{
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int gid = (blockDim.x * blockIdx.x) + tid;
    shared[tid] = -FLT_MAX;
    shared[tid + blockDim.x] = FLT_MAX;

    while (gid < N) {
        shared[tid] = max(shared[tid], d_array[gid].x);
        shared[tid + blockDim.x] = min(shared[tid + blockDim.x], d_array[gid].x);
        gid += gridDim.x * blockDim.x;
    }
    __syncthreads();
    gid = (blockDim.x * blockIdx.x) + tid;  

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s && gid < N) {
            shared[tid] = max(shared[tid], shared[tid + s]);
            shared[tid + blockDim.x] = min(shared[tid + blockDim.x], shared[tid + blockDim.x + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMaxf(d_max, shared[0]);
        atomicMinf(d_min, shared[blockDim.x]);
    }
}

__device__ __forceinline__
hipfftComplex cexpf(hipfftComplex z)
{
    hipfftComplex res; 
    float t = expf(z.x); 
    sincosf(z.y, &res.y, &res.x); 
    res.x *= t; 
    res.y *= t; 
    return res; 
}

__global__ 
void BGR2HSVKernel(uint8_t* image, int width, int height, int step) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * step + 3 * x;

        uint8_t b = image[index];
        uint8_t g = image[index + 1];
        uint8_t r = image[index + 2];

        float fB = b / 255.0f;
        float fG = g / 255.0f;
        float fR = r / 255.0f;

        float maxVal = fmaxf(fmaxf(fB, fG), fR);
        float minVal = fminf(fminf(fB, fG), fR);
        float delta = maxVal - minVal;
        float h, s, v;

        v = maxVal;

        if (maxVal > 0) {
            s = (delta / maxVal);
        } else {
            s = 0;
            h = 0;
        }

        if (s == 0) {
            h = 0;
        } else {
            if (maxVal == fR) {
                h = 60.0f * (fG - fB) / delta;
            } else if (maxVal == fG) {
                h = 120.0f + 60.0f * (fB - fR) / delta;
            } else {
                h = 240.0f + 60.0f * (fR - fG) / delta;
            }
        }

        if (h < 0) {
            h += 360;
        }

        image[index] = static_cast<uint8_t>(h / 2);
        image[index + 1] = static_cast<uint8_t>(s * 255);
        image[index + 2] = static_cast<uint8_t>(v * 255);
    }
}

__global__ void HSV2BGRKernel(uint8_t* image, int width, int height, int step) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * step + 3 * x;

        uint8_t h = image[index];
        uint8_t s = image[index + 1];
        uint8_t v = image[index + 2];

        float fH = h * 2.0f;
        float fS = s / 255.0f;
        float fV = v / 255.0f;

        int i = static_cast<int>(floor(fH / 60.0f)) % 6;
        float f = fH / 60.0f - i;
        float p = fV * (1 - fS);
        float q = fV * (1 - f * fS);
        float t = fV * (1 - (1 - f) * fS);

        switch (i) {
            case 0:
                image[index + 2] = static_cast<uint8_t>(fV * 255);
                image[index + 1] = static_cast<uint8_t>(t * 255);
                image[index] = static_cast<uint8_t>(p * 255);
                break;
            case 1:
                image[index + 2] = static_cast<uint8_t>(q * 255);
                image[index + 1] = static_cast<uint8_t>(fV * 255);
                image[index] = static_cast<uint8_t>(p * 255);
                break;
            case 2:
                image[index + 2] = static_cast<uint8_t>(p * 255);
                image[index + 1] = static_cast<uint8_t>(fV * 255);
                image[index] = static_cast<uint8_t>(t * 255);
                break;
            case 3:
                image[index + 2] = static_cast<uint8_t>(p * 255);
                image[index + 1] = static_cast<uint8_t>(q * 255);
                image[index] = static_cast<uint8_t>(fV * 255);
                break;
            case 4:
                image[index + 2] = static_cast<uint8_t>(t * 255);
                image[index + 1] = static_cast<uint8_t>(p * 255);
                image[index] = static_cast<uint8_t>(fV * 255);
                break;
            case 5:
                image[index + 2] = static_cast<uint8_t>(fV * 255);
                image[index + 1] = static_cast<uint8_t>(p * 255);
                image[index] = static_cast<uint8_t>(q * 255);
                break;
        }
    }
}

__global__
void scale_exp(hipfftComplex* d_vevid_kernel, float scalar, const size_t N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        float original = d_vevid_kernel[i].x; 
        original = original * scalar; 

        hipfftComplex temp; 
        temp.x = 0; 
        temp.y = -original; 
        
        d_vevid_kernel[i] = cexpf(temp); 
    }
}

__global__
void scale(hipfftComplex* input, float scalar, const size_t N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        input[i].x = input[i].x * scalar;
        input[i].y = input[i].y * scalar;
    }
}

__global__
void fftshift(hipfftComplex* d_vevid_kernel, const size_t width, const size_t height) 
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < width * height; i += stride) {
        if (i >= (width * height) / 2) {
            return;
        }
        else {
            if (i % width < (width / 2)) {
                hipfftComplex temp = d_vevid_kernel[i];
                d_vevid_kernel[i] = d_vevid_kernel[i + (width * (height / 2)) + (width / 2)];
                d_vevid_kernel[i + (width * (height / 2)) + (width / 2)] = temp;
            }
            else {
                hipfftComplex temp = d_vevid_kernel[i];
                d_vevid_kernel[i] = d_vevid_kernel[i + (width * (height / 2)) - (width / 2)];
                d_vevid_kernel[i + (width * (height / 2)) - (width / 2)] = temp;
            }
        }
    }
}

__global__
void populate(hipfftComplex* d_image, uint8_t* d_buffer, float b, const size_t N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        float temp = d_buffer[(i * 3) + 2];
        temp = (temp / 255.0f) + b;
        d_image[i].x = temp; 
        d_image[i].y = 0.0f; // BUG FIX HERE -- MUST RESET COMPLEX VALUE TO 0
    }
}

__global__
void hadamard(hipfftComplex* a1, hipfftComplex* a2, const size_t N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for (int i = index; i < N; i += stride) {
        hipfftComplex a1_val = a1[i];
        hipfftComplex a2_val = a2[i];
        
        // Perform complex componentwise multiplication
        hipfftComplex result;
        result.x = a1_val.x * a2_val.x - a1_val.y * a2_val.y;
        result.y = a1_val.x * a2_val.y + a1_val.y * a2_val.x;

        // Store the result back in a2
        a2[i] = result;
    }
}

__global__
void phase(hipfftComplex* vevid_image, uint8_t* image, float gain, const size_t N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        float imaginary = gain * hipCimagf(vevid_image[i]);
        float original = (float)image[(i * 3) + 2] / 255.0f;
        float temp = atan2f(imaginary, original);
        vevid_image[i].x = temp; 
    }
}

__global__
void vevid_phase(hipfftComplex* vevid_image, uint8_t* image, float gain, const size_t N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        float imaginary_approx = -gain * hipCrealf(vevid_image[i]); 
        float original = (float)image[(i * 3) + 2] / 255.0f; 
        float temp = atan2f(imaginary_approx, original); 
        vevid_image[i].x = temp; 
    }
}

__global__
void norm(hipfftComplex* d_image, uint8_t* d_buffer, float max_phase, float min_phase, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x; 
    for (int i = index; i < N; i += stride) {
        float temp = d_image[i].x; 
        temp = ((temp - min_phase) / (max_phase - min_phase)); 
        d_buffer[(i * 3) + 2] = static_cast<uint8_t>(temp * 255);
    }
}
